#include "hip/hip_runtime.h"
/***************************************************************************
 * Copyright (c) 2024 Energy Aware Runtime - Barcelona Supercomputing Center
 *
 * This program and the accompanying materials are made
 * available under the terms of the Eclipse Public License 2.0
 * which is available at https://www.eclipse.org/legal/epl-2.0/
 *
 * SPDX-License-Identifier: EPL-2.0
 **************************************************************************/

extern "C"
{
//#define SHOW_DEBUGS 1

#include <stdio.h>
#include <unistd.h>
#include <pthread.h>
#include <hip/hip_runtime.h>
#include <common/output/debug.h>
#include <common/utils/string.h>
#include <common/system/plugin_manager.h>
 
static pthread_t  thread;
static int        is_running;
static uint      *devs_running;
static uint       devs_count;
static ullong     cycles;
static ullong     run_time;
static uint       lay_time;

__global__ void kcuda_kernel(ullong cycles)
{
    long long start = clock64();
    while (clock64() < (start + cycles));
}

int kcuda_is_running()
{
    return is_running;
}

int kcuda_count_devices()
{
    int devs_count = 0;
    hipGetDeviceCount(&devs_count);
    return devs_count;
}

static void *static_kcuda_execute(void *x)
{
    static struct hipDeviceProp_t prop;
    int d;  
 
    while (1)
    {
        is_running = 1;

        for (d = 0; d < devs_count; ++d) {
            debug("Running in GPU%d? %u", d, devs_running[d]);
            if (!devs_running[d]) {
                continue;
            }
            if (hipSetDevice(d)) {
                printf("GPU%d error: %s\n", d, hipGetErrorString(hipGetLastError()));
                devs_running[d] = 0;
                continue;
            }
            if (hipGetDeviceProperties(&prop, d) != hipSuccess) {
                printf("GPU%d error: %s\n", d, hipGetErrorString(hipGetLastError()));
                devs_running[d] = 0;
                continue;
            }
            cycles = prop.clockRate * 1000LLU * run_time; 
            printf("Running GPU%d: %llu\n", d, cycles);
            kcuda_kernel<<<prop.multiProcessorCount, 64>>>(cycles);
        }
        if (hipDeviceSynchronize() != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(hipGetLastError()));
            return NULL;
        }
        hipDeviceReset();
        is_running = 0;
        sleep(lay_time);
    }

    return NULL;
}

int kcuda_execute(char **conf)
{
    run_time     = 10LLU;
    lay_time     =  2LLU; 
    devs_count   = kcuda_count_devices();
    devs_running = (uint *) calloc(devs_count, sizeof(uint));
    // Running devices
    if (conf == NULL || !rantoa(conf[0], devs_running, devs_count)) {
        // If the list is empty, we are selecting the GPU0.
        devs_running[0] = 1;
    }
    if (ARG(conf, 1)) run_time = (ullong) atoi(conf[1]);
    if (ARG(conf, 2)) lay_time = (uint)   atoi(conf[2]);
    pthread_create(&thread, NULL, static_kcuda_execute, NULL);    
    return 1;
}
}
